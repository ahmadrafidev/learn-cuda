
#include <hip/hip_runtime.h>
__global__ void vecAdd(float *a, float *b, float *c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void vecSub(float *a, float *b, float *c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] - b[idx];
    }
}

__global__ void vecDot(float *a, float *b, float *c, int N) {
    __shared__ float cache[256];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0f;
    while (idx < N) {
        temp += a[idx] * b[idx];
        idx += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}

void conjugateGradientSolver(float *A, float *b, float *x, int N, int maxIterations, float tolerance) {
    float *d_A, *d_b, *d_x, *d_r, *d_p, *d_Ap, *d_temp;
    size_t size = N * sizeof(float);
    size_t sizeA = N * N * sizeof(float);

    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_b, size);
    hipMalloc(&d_x, size);
    hipMalloc(&d_r, size);
    hipMalloc(&d_p, size);
    hipMalloc(&d_Ap, size);
    hipMalloc(&d_temp, size);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    // r = b - Ax
    vecSub<<<numBlocks, blockSize>>>(d_b, d_Ap, d_r, N);
    hipDeviceSynchronize();

    // p = r
    hipMemcpy(d_p, d_r, size, hipMemcpyDeviceToDevice);

    float rsold, rsnew;
    float *d_rsold, *d_rsnew;
    hipMalloc(&d_rsold, sizeof(float));
    hipMalloc(&d_rsnew, sizeof(float));

    vecDot<<<numBlocks, blockSize>>>(d_r, d_r, d_temp, N);
    hipDeviceSynchronize();
    hipMemcpy(&rsold, d_temp, sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < maxIterations; i++) {
        // Ap = A * p
        // Here you need to implement matrix-vector multiplication for A * p

        // alpha = rsold / (p' * Ap)
        vecDot<<<numBlocks, blockSize>>>(d_p, d_Ap, d_temp, N);
        hipDeviceSynchronize();
        float pAp;
        hipMemcpy(&pAp, d_temp, sizeof(float), hipMemcpyDeviceToHost);
        float alpha = rsold / pAp;

        // x = x + alpha * p
        // r = r - alpha * Ap
        // rsnew = r' * r

        vecDot<<<numBlocks, blockSize>>>(d_r, d_r, d_temp, N);
        hipDeviceSynchronize();
        hipMemcpy(&rsnew, d_temp, sizeof(float), hipMemcpyDeviceToHost);

        if (sqrt(rsnew) < tolerance) {
            break;
        }

        // p = r + (rsnew / rsold) * p
        rsold = rsnew;
    }

    hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ap);
    hipFree(d_temp);
    hipFree(d_rsold);
    hipFree(d_rsnew);
}
