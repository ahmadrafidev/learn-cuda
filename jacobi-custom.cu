#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define MAX_ITER 1000 // jumlah iterasi maksimum
#define TOLERANCE 1e-6 // toleransi konvergensi

__global__ void jacobiIterationKernel(float *A, float *b, float *x, float *x_new, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float sigma = 0.0;
        for (int j = 0; j < n; j++) {
            if (j != idx) {
                sigma += A[idx * n + j] * x[j];
            }
        }
        if (A[idx * n + idx] != 0) {
            x_new[idx] = (b[idx] - sigma) / A[idx * n + idx];
        } else {
            x_new[idx] = x[idx]; // Hindari pembagian dengan nol
        }
    }
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

bool checkConvergence(float *x, float *x_new, int n, float tolerance) {
    for (int i = 0; i < n; i++) {
        if (fabs(x[i] - x_new[i]) > tolerance) {
            return false;
        }
    }
    return true;
}

void jacobiIteration(float *A, float *b, float *x, int n) {
    float *d_A, *d_b, *d_x, *d_x_new;
    size_t size = n * n * sizeof(float);
    hipError_t err;

    err = hipMalloc(&d_A, size);
    checkCudaError(err, "Failed to allocate device memory for A");

    err = hipMalloc(&d_b, n * sizeof(float));
    checkCudaError(err, "Failed to allocate device memory for b");

    err = hipMalloc(&d_x, n * sizeof(float));
    checkCudaError(err, "Failed to allocate device memory for x");

    err = hipMalloc(&d_x_new, n * sizeof(float));
    checkCudaError(err, "Failed to allocate device memory for x_new");

    err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    checkCudaError(err, "Failed to copy matrix A to device");

    err = hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);
    checkCudaError(err, "Failed to copy vector b to device");

    err = hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    checkCudaError(err, "Failed to copy vector x to device");

    dim3 blockSize(256);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x);

    float *h_x_new = (float *)malloc(n * sizeof(float));

    for (int iter = 0; iter < MAX_ITER; iter++) {
        jacobiIterationKernel<<<gridSize, blockSize>>>(d_A, d_b, d_x, d_x_new, n);
        err = hipGetLastError();
        checkCudaError(err, "Kernel execution failed");

        err = hipMemcpy(h_x_new, d_x_new, n * sizeof(float), hipMemcpyDeviceToHost);
        checkCudaError(err, "Failed to copy result x_new to host");

        // Print values for debugging
        printf("Iteration %d: ", iter);
        for (int i = 0; i < n; i++) {
            printf("%f ", h_x_new[i]);
        }
        printf("\n");

        // Check for NaN or Inf
        for (int i = 0; i < n; i++) {
            if (isnan(h_x_new[i]) || isinf(h_x_new[i])) {
                printf("NaN or Inf detected at index %d\n", i);
                free(h_x_new);
                hipFree(d_A);
                hipFree(d_b);
                hipFree(d_x);
                hipFree(d_x_new);
                return; // Stop execution
            }
        }

        // Check convergence
        if (checkConvergence(x, h_x_new, n, TOLERANCE)) {
            printf("Converged at iteration %d\n", iter);
            break;
        }

        // Copy new values to x for next iteration
        err = hipMemcpy(d_x, d_x_new, n * sizeof(float), hipMemcpyDeviceToDevice);
        checkCudaError(err, "Failed to copy x_new to x on device");

        // Update host x for next iteration
        for (int i = 0; i < n; i++) {
            x[i] = h_x_new[i];
        }
    }

    err = hipMemcpy(x, d_x_new, n * sizeof(float), hipMemcpyDeviceToHost);
    checkCudaError(err, "Failed to copy final result x to host");

    free(h_x_new);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_x);
    hipFree(d_x_new);
}

int main(int argc, char *argv[]) {
    int n = 1024; // Default size
    if (argc > 1) {
        n = atoi(argv[1]);
    }

    float *A = (float *)malloc(n * n * sizeof(float));
    float *b = (float *)malloc(n * sizeof(float));
    float *x = (float *)malloc(n * sizeof(float));

    // Inisialisasi matriks A dan vektor b
    for (int i = 0; i < n; i++) {
        b[i] = (float)(i + 1); // Nilai yang lebih beragam untuk b
        x[i] = 0.0; // Inisialisasi dengan nilai nol
        for (int j = 0; j < n; j++) {
            if (i == j) {
                A[i * n + j] = (float)(n); // Diagonal dominan
            } else {
                A[i * n + j] = 0.1; // Nilai lebih kecil untuk elemen non-diagonal
            }
        }
    }

    jacobiIteration(A, b, x, n);

    // Cetak beberapa elemen hasil x
    printf("Hasil vektor x:\n");
    for (int i = 0; i < n; i++) {
        printf("%f\n", x[i]);
    }

    free(A);
    free(b);
    free(x);

    return 0;
}
