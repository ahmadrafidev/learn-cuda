
#include <hip/hip_runtime.h>
__global__ void matrixMultiplySharedKernel(float *A, float *B, float *C, int N) {
    __shared__ float s_A[32][32];
    __shared__ float s_B[32][32];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    for (int k = 0; k < (N + 32 - 1) / 32; k++) {
        if (k * 32 + threadIdx.x < N && row < N) {
            s_A[threadIdx.y][threadIdx.x] = A[row * N + k * 32 + threadIdx.x];
        } else {
            s_A[threadIdx.y][threadIdx.x] = 0.0;
        }
        if (k * 32 + threadIdx.y < N && col < N) {
            s_B[threadIdx.y][threadIdx.x] = B[(k * 32 + threadIdx.y) * N + col];
        } else {
            s_B[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads();

        for (int n = 0; n < 32; ++n) {
            sum += s_A[threadIdx.y][n] * s_B[n][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < N && col < N) {
        C[row * N + col] = sum;
    }
}

void parallelMatrixMultiplyShared(float *A, float *B, float *C, int N, int blockSize) {
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid((N + blockSize - 1) / blockSize, (N + blockSize - 1) / blockSize);

    matrixMultiplySharedKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int N = 1024; // Change as needed
    int blockSize = 32; // Change as needed
    float *A = (float*)malloc(N * N * sizeof(float));
    float *B = (float*)malloc(N * N * sizeof(float));
    float *C = (float*)malloc(N * N * sizeof(float));

    // Initialize matrices A and B
    for (int i = 0; i < N * N; i++) {
        A[i] = rand() % 100;
        B[i] = rand() % 100;
    }

    parallelMatrixMultiplyShared(A, B, C, N, blockSize);

    // Print a small part of the matrix to verify correctness
    for (int i = 0; i < 5; i++) {
        for (int j = 0; j < 5; j++) {
            printf("%f ", C[i * N + j]);
        }
        printf("\n");
    }

    free(A);
    free(B);
    free(C);

    return 0;
}
