#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void matrixMultiplyKernel(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

void parallelMatrixMultiply(float *A, float *B, float *C, int N, int blockSize) {
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid((N + blockSize - 1) / blockSize, (N + blockSize - 1) / blockSize);

    matrixMultiplyKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int N = 1024; // Change as needed
    int blockSize = 16; // Change as needed
    float *A = (float*)malloc(N * N * sizeof(float));
    float *B = (float*)malloc(N * N * sizeof(float));
    float *C = (float*)malloc(N * N * sizeof(float));

    // Initialize matrices A and B
    for (int i = 0; i < N * N; i++) {
        A[i] = rand() % 100;
        B[i] = rand() % 100;
    }

    parallelMatrixMultiply(A, B, C, N, blockSize);

    // Print a small part of the matrix to verify correctness
    for (int i = 0; i < 5; i++) {
        for (int j = 0; j < 5; j++) {
            printf("%f ", C[i * N + j]);
        }
        printf("\n");
    }

    free(A);
    free(B);
    free(C);

    return 0;
}
